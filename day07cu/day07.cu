#include "hip/hip_runtime.h"
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/remove.h>

__device__ inline void int_to_string(int64_t value, char* str) {
    char* ptr = str;
    if (value == 0) {
        *ptr++ = '0';
    } else {
        if (value < 0) {
            *ptr++ = '-';
            value = -value;
        }
        char* start = ptr;
        while (value) {
            *ptr++ = '0' + (value % 10);
            value /= 10;
        }
        *ptr = '\0';
        // Reverse the string
        char* end = ptr - 1;
        while (start < end) {
            char temp = *start;
            *start++ = *end;
            *end-- = temp;
        }
    }
    *ptr = '\0';
}

__device__ inline int64_t string_to_int(const char* str) {
    int64_t result = 0;
    while (*str) {
        result = result * 10 + (*str - '0');
        str++;
    }
    return result;
}

__device__ inline size_t cu_strlen(const char* str) {
    size_t len = 0;
    while (*str++) {
        len++;
    }
    return len;
}

__device__ inline int64_t concatenate_integers(int64_t a, int64_t b) {
    char str[64] = {0}; // Initialize to ensure it's empty
    int_to_string(a, str);
    int_to_string(b, str + cu_strlen(str)); // Append b to the end of the string representation of a
    return string_to_int(str);
}

struct is_zero
{
    __host__ __device__
    bool operator()(const int64_t x) const
    {
        return x == 0;
    }
};



__global__ void calc_state_kernel(
    int32_t total_combinations, 
    int64_t* numbers, int32_t* cum_number_lengths,
    int64_t* results, int32_t* cum_combinations,
    int64_t* ret) {
    
    int32_t idx_combination = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx_combination >= total_combinations) return;

    int32_t idx_line = 0;
    while(true) {
        if (cum_combinations[idx_line] > idx_combination) break;
        idx_line++;
    }

    int32_t offset_idx_numbers = 0;
    int32_t offset_idx_combinations = 0;

    if (idx_line > 0) {
        offset_idx_numbers = cum_number_lengths[idx_line - 1];
        offset_idx_combinations = cum_combinations[idx_line - 1];
    }

    auto result = results[idx_line];

    int32_t state = idx_combination - offset_idx_combinations;
    int32_t in = offset_idx_numbers;
    int64_t intermediate = numbers[in];
    for (in = in + 1; in < cum_number_lengths[idx_line]; in++) {
        int64_t number = numbers[in];

        if (intermediate > result) {
            ret[idx_combination] = 0;
            return;
        }

        auto op = state & 1;
        if (op) {
            intermediate += number;
        } else {
            intermediate *= number;
        }
        state = state>>1;
    }
    if (intermediate == result) {
        ret[idx_combination] = result;
    } else {
        ret[idx_combination] = 0;
    }
    return;
}

__global__ void calc_state_kernel2(
    int64_t total_combinations, 
    int64_t* numbers, int32_t* cum_number_lengths,
    int64_t* results, int32_t* cum_combinations,
    int64_t* ret) {
    int32_t idx_combination = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx_combination >= total_combinations) {
        return;
    }

    int32_t idx_line = 0;
    while(true) {
        if (cum_combinations[idx_line] > idx_combination) {
            break;
        }
        idx_line++;
    }

    int32_t offset_idx_numbers = 0;
    int32_t offset_idx_combinations = 0;

    if (idx_line > 0) {
        offset_idx_numbers = cum_number_lengths[idx_line - 1];
        offset_idx_combinations = cum_combinations[idx_line - 1];
    }

    auto result = results[idx_line];

    int32_t state = idx_combination - offset_idx_combinations;
    int32_t in = offset_idx_numbers;
    int64_t intermediate = numbers[in];
    for (in = in + 1; in < cum_number_lengths[idx_line]; in++) {
        int64_t number = numbers[in];

        if (intermediate > result) {
            ret[idx_combination] = 0;
            return;
        }

        auto op = state % 3;
        if (op==0) {
            intermediate += number;
        } else if (op==1) {
            intermediate *= number;
        } else {
            intermediate = concatenate_integers(intermediate, number);
        }

        state = state / 3;
    }
    if (intermediate == result) {
        ret[idx_combination] = result;
    } else {
        ret[idx_combination] = 0;
    }
    return;
}

int64_t calc_states(
    const std::vector<int64_t>& numbers, std::vector<int32_t>& cum_number_lengths, 
    const std::vector<int64_t>& results, std::vector<int32_t>& cum_number_of_combinations,
    int part) {
    
    // Copy data to device
    thrust::device_vector<int64_t> d_numbers(numbers);
    int64_t* d_numbers_ptr = thrust::raw_pointer_cast(d_numbers.data());

    thrust::device_vector<int32_t> d_cum_number_lengths(cum_number_lengths);
    int32_t* d_cum_number_lengths_ptr = thrust::raw_pointer_cast(d_cum_number_lengths.data());
    
    thrust::device_vector<int32_t> d_cum_combinations(cum_number_of_combinations);
    int32_t* d_cum_combinations_ptr = thrust::raw_pointer_cast(d_cum_combinations.data());

    thrust::device_vector<int64_t> d_results(results);
    int64_t* d_results_ptr = thrust::raw_pointer_cast(d_results.data());

    // get the number of threads launched
    int64_t total_combinations = cum_number_of_combinations.back();

    // prepare return vector
    thrust::device_vector<int64_t> d_return(total_combinations);
    int64_t* d_ret_ptr = thrust::raw_pointer_cast(d_return.data());

    int block_size = 1024;
    int grid_size = (total_combinations + block_size - 1) / block_size;
    if (part == 1) {
        calc_state_kernel<<<grid_size, block_size>>>(
            total_combinations, 
            d_numbers_ptr, d_cum_number_lengths_ptr, 
            d_results_ptr, d_cum_combinations_ptr, d_ret_ptr);
    } else {
        calc_state_kernel2<<<grid_size, block_size>>>(
            total_combinations, 
            d_numbers_ptr, d_cum_number_lengths_ptr, 
            d_results_ptr, d_cum_combinations_ptr, d_ret_ptr);
    }
    hipDeviceSynchronize();
    
    // Remove zeros
    auto end = thrust::remove_if(d_return.begin(), d_return.end(), is_zero());
    d_return.erase(end, d_return.end());
    // Remove duplicates
    end = thrust::unique(d_return.begin(), d_return.end());
    d_return.erase(end, d_return.end());
    // Sum
    int64_t sum = thrust::reduce(d_return.begin(), d_return.end(), int64_t(0));

    return sum;
}