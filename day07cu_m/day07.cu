#include "hip/hip_runtime.h"
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/remove.h>

__device__ inline void int_to_string(int64_t value, char* str) {
    char* ptr = str;
    if (value == 0) {
        *ptr++ = '0';
    } else {
        if (value < 0) {
            *ptr++ = '-';
            value = -value;
        }
        char* start = ptr;
        while (value) {
            *ptr++ = '0' + (value % 10);
            value /= 10;
        }
        *ptr = '\0';
        // Reverse the string
        char* end = ptr - 1;
        while (start < end) {
            char temp = *start;
            *start++ = *end;
            *end-- = temp;
        }
    }
    *ptr = '\0';
}

__device__ inline int64_t string_to_int(const char* str) {
    int64_t result = 0;
    while (*str) {
        result = result * 10 + (*str - '0');
        str++;
    }
    return result;
}

__device__ inline size_t cu_strlen(const char* str) {
    size_t len = 0;
    while (*str++) {
        len++;
    }
    return len;
}

__device__ inline int64_t concatenate_integers(int64_t a, int64_t b) {
    char str[64] = {0}; // Initialize to ensure it's empty
    int_to_string(a, str);
    int_to_string(b, str + cu_strlen(str)); // Append b to the end of the string representation of a
    return string_to_int(str);
}

struct is_zero
{
    __host__ __device__
    bool operator()(const int64_t x) const
    {
        return x == 0;
    }
};


__global__ void calc_state_kernel(
    int64_t* numbers, int32_t numbers_size, int64_t result, int32_t combinations, int64_t* ret, int32_t return_offset) {
    
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= combinations) return;

    int32_t state = idx;
    int32_t return_idx = return_offset + idx;
    int64_t intermediate = numbers[0];
    for (auto i = 1; i < numbers_size; i++) {
        int64_t number = numbers[i];

        if (intermediate > result) {
            ret[return_idx] = 0;
            return;
        }

        auto op = state & 1;
        if (op) {
            intermediate += number;
        } else {
            intermediate *= number;
        }
        state = state>>1;
    }
    if (intermediate == result) {
        ret[return_idx] = result;
    } else {
        ret[return_idx] = 0;
    }
    return;
}

__global__ void calc_state_kernel2(
    int64_t* numbers, int32_t numbers_size, int64_t result, int32_t combinations, int64_t* ret, int32_t return_offset) {
    
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= combinations) return;

    int32_t state = idx;
    int32_t return_idx = return_offset + idx;
    int64_t intermediate = numbers[0];
    for (auto i = 1; i < numbers_size; i++) {
        int64_t number = numbers[i];

        if (intermediate > result) {
            ret[return_idx] = 0;
            return;
        }

        auto op = state % 3;
        if (op==0) {
            intermediate += number;
        } else if (op==1) {
            intermediate *= number;
        } else {
            intermediate = concatenate_integers(intermediate, number);
        }
        state = state / 3;
    }
    if (intermediate == result) {
        ret[return_idx] = result;
    } else {
        ret[return_idx] = 0;
    }
    return;
}

int64_t calc_states(
    const std::vector<std::vector<int64_t>>& numbers, 
    const std::vector<int64_t>& results,
    const std::vector<int32_t>& number_of_combinations,
    int32_t total_combinations, int32_t part) {
    
    // prepare return vector
    thrust::device_vector<int64_t> d_return(total_combinations);
    int64_t* d_return_ptr = thrust::raw_pointer_cast(d_return.data());

    int32_t return_idx_offset = 0;
    for (size_t i = 0; i < numbers.size(); i++) {
        // Copy data to device
        thrust::device_vector<int64_t> d_numbers(numbers[i]);
        int64_t* d_numbers_ptr = thrust::raw_pointer_cast(d_numbers.data());
        int32_t numbers_size = numbers[i].size();

        int64_t result = results[i];
        int32_t combinations = number_of_combinations[i];

        int block_size = 1024;
        int grid_size = (combinations + block_size - 1) / block_size;
        if (part == 1) {
            calc_state_kernel<<<grid_size, block_size>>>(
                d_numbers_ptr, numbers_size, result, combinations, d_return_ptr, return_idx_offset);
        } else {
            calc_state_kernel2<<<grid_size, block_size>>>(
                d_numbers_ptr, numbers_size, result, combinations, d_return_ptr, return_idx_offset);
        }
        return_idx_offset += combinations;
    }
    hipDeviceSynchronize();
    
    // Remove zeros
    auto end = thrust::remove_if(d_return.begin(), d_return.end(), is_zero());
    d_return.erase(end, d_return.end());
    // Remove duplicates
    end = thrust::unique(d_return.begin(), d_return.end());
    d_return.erase(end, d_return.end());
    // Sum
    int64_t sum = thrust::reduce(d_return.begin(), d_return.end(), int64_t(0));

    return sum;
}