#include "hip/hip_runtime.h"
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/remove.h>

__device__ void int_to_string(int64_t value, char* str) {
    char* ptr = str;
    if (value == 0) {
        *ptr++ = '0';
    } else {
        if (value < 0) {
            *ptr++ = '-';
            value = -value;
        }
        char* start = ptr;
        while (value) {
            *ptr++ = '0' + (value % 10);
            value /= 10;
        }
        *ptr = '\0';
        // Reverse the string
        char* end = ptr - 1;
        while (start < end) {
            char temp = *start;
            *start++ = *end;
            *end-- = temp;
        }
    }
    *ptr = '\0';
}

__device__ int64_t string_to_int(const char* str) {
    int64_t result = 0;
    while (*str) {
        result = result * 10 + (*str - '0');
        str++;
    }
    return result;
}

__device__ size_t cu_strlen(const char* str) {
    size_t len = 0;
    while (*str++) {
        len++;
    }
    return len;
}

__device__ int64_t concatenate_integers(int64_t a, int64_t b) {
    char str[64] = {0}; // Initialize to ensure it's empty
    int_to_string(a, str);
    int_to_string(b, str + cu_strlen(str)); // Append b to the end of the string representation of a
    return string_to_int(str);
}

struct is_zero
{
    __host__ __device__
    bool operator()(const int64_t x) const
    {
        return x == 0;
    }
};


__global__ void calc_state_kernel(
    int32_t total_combinations, 
    int64_t* numbers, int32_t* cum_number_lengths,
    int64_t* results, int32_t* cum_combinations,
    int64_t* ret) {
    int32_t idx_combination = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx_combination >= total_combinations) {
        return;
    }

    int32_t idx_line = 0;
    while(true) {
        if (cum_combinations[idx_line] > idx_combination) {
            break;
        }
        idx_line++;
    }

    int32_t offset_idx_numbers = 0;
    int32_t offset_idx_combinations = 0;

    if (idx_line > 0) {
        offset_idx_numbers = cum_number_lengths[idx_line - 1];
        offset_idx_combinations = cum_combinations[idx_line - 1];
    }

    auto result = results[idx_line];

    int32_t state = idx_combination - offset_idx_combinations;
    int32_t in = offset_idx_numbers;
    int64_t intermediate = numbers[in];
    for (in = in + 1; in < cum_number_lengths[idx_line]; in++) {
        int64_t number = numbers[in];

        if (intermediate > result) {
            ret[idx_combination] = 0;
            return;
        }

        auto op = state & 1;
        if (op) {
            intermediate += number;
        } else {
            intermediate *= number;
        }
        state = state>>1;
    }
    if (intermediate == result) {
        ret[idx_combination] = result;
    } else {
        ret[idx_combination] = 0;
    }
    return;
}

__global__ void calc_state_kernel2(
    int64_t total_combinations, 
    int64_t* numbers, int32_t* cum_number_lengths,
    int64_t* results, int32_t* cum_combinations,
    int64_t* ret) {
    int32_t idx_combination = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx_combination >= total_combinations) {
        return;
    }

    int32_t idx_line = 0;
    while(true) {
        if (cum_combinations[idx_line] > idx_combination) {
            break;
        }
        idx_line++;
    }

    int32_t offset_idx_numbers = 0;
    int32_t offset_idx_combinations = 0;

    if (idx_line > 0) {
        offset_idx_numbers = cum_number_lengths[idx_line - 1];
        offset_idx_combinations = cum_combinations[idx_line - 1];
    }

    auto result = results[idx_line];

    int32_t state = idx_combination - offset_idx_combinations;
    int32_t in = offset_idx_numbers;
    int64_t intermediate = numbers[in];
    for (in = in + 1; in < cum_number_lengths[idx_line]; in++) {
        int64_t number = numbers[in];

        if (intermediate > result) {
            ret[idx_combination] = 0;
            return;
        }

        auto op = state % 3;
        if (op==0) {
            intermediate += number;
        } else if (op==1) {
            intermediate *= number;
        } else {
            intermediate = concatenate_integers(intermediate, number);
        }

        state = state / 3;
    }
    if (intermediate == result) {
        ret[idx_combination] = result;
    } else {
        ret[idx_combination] = 0;
    }
    return;
}

template<typename T>
T* copy_vector_to_gpu(const std::vector<T>& vec) {
    thrust::device_vector<T> d_vec(vec);
    return thrust::raw_pointer_cast(d_vec.data());
}

int64_t calc_states(
    const std::vector<int64_t>& numbers, std::vector<int32_t>& number_lengths, 
    const std::vector<int64_t>& results, std::vector<int32_t>& number_of_combinations,
    int part) {
    
    auto d_numbers_ptr = copy_vector_to_gpu(numbers);

    // cumulative sum of numbers length to get number offset from line idx
    thrust::inclusive_scan(number_lengths.begin(), number_lengths.end(), number_lengths.begin());
    thrust::device_vector<int32_t> d_number_lengths(number_lengths);
    int32_t* d_number_lengths_ptr = thrust::raw_pointer_cast(d_number_lengths.data());
    
    // cumulative sum of number of combinations to get line idx from combination idx
    thrust::inclusive_scan(number_of_combinations.begin(), number_of_combinations.end(), number_of_combinations.begin());
    thrust::device_vector<int32_t> d_combinations(number_of_combinations);
    int32_t* d_combinations_ptr = thrust::raw_pointer_cast(d_combinations.data());


    thrust::device_vector<int64_t> d_results(results);
    int64_t* d_results_ptr = thrust::raw_pointer_cast(d_results.data());

    // get the number of threads launched
    int64_t total_combinations = number_of_combinations.back();

    // prepare return vector
    thrust::device_vector<int64_t> d_ret(total_combinations);
    int64_t* d_ret_ptr = thrust::raw_pointer_cast(d_ret.data());

    int block_size = 1024;
    int grid_size = (total_combinations + block_size - 1) / block_size;
    if (part == 1) {
        calc_state_kernel<<<grid_size, block_size>>>(total_combinations, d_numbers_ptr, d_number_lengths_ptr, d_results_ptr, d_combinations_ptr, d_ret_ptr);
    } else {
        calc_state_kernel2<<<grid_size, block_size>>>(total_combinations, d_numbers_ptr, d_number_lengths_ptr, d_results_ptr, d_combinations_ptr, d_ret_ptr);
    }
    hipDeviceSynchronize();
    
    // Remove zeros
    auto end = thrust::remove_if(d_ret.begin(), d_ret.end(), is_zero());
    d_ret.erase(end, d_ret.end());
    // Remove duplicates
    end = thrust::unique(d_ret.begin(), d_ret.end());
    d_ret.erase(end, d_ret.end());

    int64_t sum = thrust::reduce(d_ret.begin(), d_ret.end(), int64_t(0));

    return sum;
}